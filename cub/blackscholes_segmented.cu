#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <cstdlib>
#include <sys/time.h>
#include <unistd.h>

using namespace std;

#define cudaSucceeded(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) {
    std::cerr << "cudaAssert failed: "
              << hipGetErrorString(code)
              << file << ":" << line
              << std::endl;
    if (abort) {
      exit(code);
    }
  }
}

static struct timeval t_start, t_end;

void start_timing() {
  cudaSucceeded(hipDeviceSynchronize());
  gettimeofday(&t_start, NULL);
}

void end_timing() {
  cudaSucceeded(hipDeviceSynchronize());
  gettimeofday(&t_end, NULL);
}

int get_us() {
  return (t_end.tv_sec*1000000+t_end.tv_usec) - (t_start.tv_sec*1000000+t_start.tv_usec);
}

class make_option {
private:
  __host__ __device__ static
  float horner(float x) {
    float c1 = 0.31938153, c2 = -0.356563782, c3 = 1.781477937, c4 = -1.821255978, c5 = 1.330274429;
    return x * (c1 + x * (c2 + x * (c3 + x * (c4 + x * c5))));
  }


  __host__ __device__ static
  float cnd0(float d) {
    float k        = 1.0 / (1.0 + 0.2316419 * abs(d));
    float p        = horner(k);
    float rsqrt2pi = 0.39894228040143267793994605993438;
    return rsqrt2pi * exp(-0.5*d*d) * p;
  }

  __host__ __device__ static
  float cnd(float d) {
    float c = cnd0(d);
    return 0.0 < d ? 1.0 - c : c;
  }

  int i;
  const double *d_rs;
  const double *d_vs;
  const int days;

  typedef make_option self_type;

public:
  __host__ __device__
  make_option(int i, const double *d_rs, const double *d_vs, int days) :
    i(i), d_rs(d_rs), d_vs(d_vs), days(days) {}

  typedef std::random_access_iterator_tag iterator_category;
  typedef double value_type;
  typedef int difference_type;
  typedef double* pointer;
  typedef double reference;

  __host__ __device__
  double value_at(int i) const {
    int option = i / days;
    int day = i % days;
    double r = d_rs[option];
    double v = d_vs[option];

    bool call = day % 2 == 0;

    double price = 58 + 5 * (1+day)/double(days);
    double strike = 65;
    double years = (1+day)/365.0;
    double v_sqrtT = v * sqrt(years);
    double d1      = (log(price / strike) + (r + 0.5 * v * v) * years) / v_sqrtT;
    double d2      = d1 - v_sqrtT;
    double cndD1   = cnd(d1);
    double cndD2   = cnd(d2);
    double x_expRT = strike * exp(-r * years);

    if (call) {
      return price * cndD1 - x_expRT * cndD2;
    } else {
      return x_expRT * (1.0 - cndD2) - price * (1.0 - cndD1);
    }
  }


  __device__
  double operator*() const {
    return value_at(i);
  }

  __host__ __device__ self_type operator++(int)
  {
    self_type retval = *this;
    i++;
    return retval;
  }

  __host__ __device__ __forceinline__ self_type operator++()
  {
    i++;
    return *this;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator+(Distance n) const
  {
    self_type retval(i + int(n), d_rs, d_vs, days);
    return retval;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type& operator+=(Distance n)
  {
    i += (int) n;
    return *this;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator-(Distance n) const
  {
    self_type retval(i - (int)n, d_rs, d_vs, days);
    return retval;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type& operator-=(Distance n)
  {
    i -= n;
    return *this;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ reference operator[](Distance n) const
  {
    return value_at(i+(int)n);
  }
};

int main(int argc, char** argv) {

  int num_segments = pow(2,atoi(argv[1]));
  int segment_size = pow(2,atoi(argv[2]));

  cerr << num_segments << " segments of " << segment_size << " elements each" << endl;

  int *h_offsets = new int[num_segments+1];

  for (int i = 0; i < num_segments+1; i++) {
    h_offsets[i] = i * segment_size;
  }

  double *h_rs = new double[num_segments];
  double *h_vs = new double[num_segments];

  srand(31337);

  for (int i = 0; i < num_segments; i++) {
    h_rs[i] = rand()/double(RAND_MAX);
    h_vs[i] = rand()/double(RAND_MAX);
  }

  int *d_offsets;
  double *d_rs;
  double *d_vs;
  int *d_out;
  cudaSucceeded(hipMalloc(&d_offsets, (num_segments+1)*sizeof(int)));
  cudaSucceeded(hipMalloc(&d_rs, num_segments*sizeof(double)));
  cudaSucceeded(hipMalloc(&d_vs, num_segments*sizeof(double)));
  cudaSucceeded(hipMalloc(&d_out, num_segments*sizeof(double)));

  cudaSucceeded(hipMemcpy(d_offsets, h_offsets, (num_segments+1)*sizeof(int),
                           hipMemcpyHostToDevice));
  cudaSucceeded(hipMemcpy(d_rs, h_rs, num_segments*sizeof(double),
                           hipMemcpyHostToDevice));
  cudaSucceeded(hipMemcpy(d_vs, h_vs, num_segments*sizeof(double),
                           hipMemcpyHostToDevice));
  cudaSucceeded(hipDeviceSynchronize());

  void     *d_temp_storage = NULL;

  // Now time.

  static const int num_runs = 100;
  int total_us = 0;

  // We re-allocate memory for the output and intermediary arrays,
  // because that is also what the Futhark-generated code does
  // (including the computation of how much to allocate).
  for (int i = 0; i < num_runs; i++) {
    cudaSucceeded(hipFree(d_out));
    cudaSucceeded(hipFree(d_temp_storage));

    start_timing();
    cudaSucceeded(hipMalloc(&d_out, num_segments*sizeof(int)));

    size_t temp_storage_bytes = 0;
    cudaSucceeded(hipcub::DeviceSegmentedReduce::Sum
                  (d_temp_storage, temp_storage_bytes,
                   make_option(0, d_rs, d_vs, segment_size), d_out,
                   num_segments,
                   d_offsets, d_offsets + 1));

    cudaSucceeded(hipMalloc(&d_temp_storage, temp_storage_bytes));

    cudaSucceeded(hipcub::DeviceSegmentedReduce::Sum
                  (d_temp_storage, temp_storage_bytes,
                   make_option(0, d_rs, d_vs, segment_size), d_out,
                   num_segments,
                   d_offsets, d_offsets + 1));
    end_timing();
    total_us += get_us();
  }
  cerr << total_us/num_runs << "us" << endl;
  if (!isatty(1)) {
    cout << total_us/num_runs;
  }

  int * h_out = new int[num_segments];
  cudaSucceeded(hipMemcpy(h_out, d_out, num_segments*sizeof(int),
                           hipMemcpyDeviceToHost));

  // No validation; trust CUB.

  return 0;
}
