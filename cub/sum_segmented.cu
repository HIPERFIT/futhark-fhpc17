#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <cstdlib>
#include <sys/time.h>
#include <unistd.h>

using namespace std;

#define cudaSucceeded(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) {
    std::cerr << "cudaAssert failed: "
              << hipGetErrorString(code)
              << file << ":" << line
              << std::endl;
    if (abort) {
      exit(code);
    }
  }
}

static struct timeval t_start, t_end;

void start_timing() {
  cudaSucceeded(hipDeviceSynchronize());
  gettimeofday(&t_start, NULL);
}

void end_timing() {
  cudaSucceeded(hipDeviceSynchronize());
  gettimeofday(&t_end, NULL);
}

int get_us() {
  return (t_end.tv_sec*1000000+t_end.tv_usec) - (t_start.tv_sec*1000000+t_start.tv_usec);
}

int main(int argc, char** argv) {

  int num_segments = pow(2,atoi(argv[1]));
  int segment_size = pow(2,atoi(argv[2]));
  int num_elements = num_segments * segment_size;

  cerr << num_segments << " segments of " << segment_size << " elements each" << endl;

  int *h_offsets = new int[num_segments+1];

  for (int i = 0; i < num_segments+1; i++) {
    h_offsets[i] = i * segment_size;
  }

  int *h_in = new int[num_elements];

  for (int i = 0; i < num_elements; i++) {
    h_in[i] = i % segment_size;
  }

  int *d_offsets;
  int *d_in;
  int *d_out;
  cudaSucceeded(hipMalloc(&d_offsets, (num_segments+1)*sizeof(int)));
  cudaSucceeded(hipMalloc(&d_in, num_elements*sizeof(int)));
  cudaSucceeded(hipMalloc(&d_out, num_segments*sizeof(int)));

  cudaSucceeded(hipMemcpy(d_offsets, h_offsets, (num_segments+1)*sizeof(int),
                           hipMemcpyHostToDevice));
  cudaSucceeded(hipMemcpy(d_in, h_in, num_elements*sizeof(int),
                           hipMemcpyHostToDevice));
  cudaSucceeded(hipDeviceSynchronize());

  void     *d_temp_storage = NULL;

  // Now time.

  static const int num_runs = 100;
  int total_us = 0;

  // We re-allocate memory for the output and intermediary arrays,
  // because that is also what the Futhark-generated code does
  // (including the computation of how much to allocate).
  for (int i = 0; i < num_runs; i++) {
    cudaSucceeded(hipFree(d_out));
    cudaSucceeded(hipFree(d_temp_storage));

    start_timing();
    cudaSucceeded(hipMalloc(&d_out, num_segments*sizeof(int)));

    size_t temp_storage_bytes = 0;
    cudaSucceeded(hipcub::DeviceSegmentedReduce::Sum
                  (d_temp_storage, temp_storage_bytes, d_in, d_out,
                   num_segments,
                   d_offsets, d_offsets + 1));

    cudaSucceeded(hipMalloc(&d_temp_storage, temp_storage_bytes));

    cudaSucceeded(hipcub::DeviceSegmentedReduce::Sum
                  (d_temp_storage, temp_storage_bytes, d_in, d_out,
                   num_segments,
                   d_offsets, d_offsets + 1));
    end_timing();
    total_us += get_us();
  }
  cerr << total_us/num_runs << "us" << endl;
  if (!isatty(1)) {
    cout << total_us/num_runs;
  }

  int * h_out = new int[num_segments];
  cudaSucceeded(hipMemcpy(h_out, d_out, num_segments*sizeof(int),
                           hipMemcpyDeviceToHost));

  int correct_segment_sum = 0;
  for (int i = 0; i < segment_size; i++) {
    correct_segment_sum += h_in[i];
  }

  for (int i = 0; i < num_segments; i++) {
    if (h_out[i] != correct_segment_sum) {
      cerr << "segment " << i << " has sum " << h_out[i]
           << "(expected " << correct_segment_sum << ")" << endl;
    }
  }

  return 0;
}
